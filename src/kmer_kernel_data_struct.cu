#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "tipos_data_struct.h"



//Compute k-mer index
__global__ void ComputeFrequency(char *Seq, struct counter *d_counter, lint *d_start, int *d_length, const int k, lint nN, ushort offset, int n_sequence, int n_combination)
{
   int idx =  blockIdx.x;

   int start = idx * offset;
   int end   = start + offset;

   for(lint id = start; id < end; id++)
   {
      
      int index = -1;
      if (id < nN)
      {
         lint id_sequence;
         lint p;
         for (p = 0; p < n_sequence; p++)
         {
            if(d_start[p] < id && id < (d_start[p] + d_length[p]))
            {
               id_sequence = p;
            }
         }
         for( lint i = 0; i < k; i++ )
         {
            char nuc = Seq[i + id];
            if (nuc != -1) //Verifica se ha alguem que nao deve ser processado
            {
               index += nuc * powf(4, ((k-1)-i));
            }
            else
            {
               index = -1;
               break;
            }
         }//End for i

         if(index != -1)
         {
            __threadfence();
            for (int t = 0; t < n_combination; t++){
                if (d_counter[id_sequence].index[t] == -1){
                    atomicAdd(&d_counter[id_sequence].index[t], index);// Value of the combination
                    atomicAdd(&d_counter[id_sequence].frequency[t], 1);// Value of the combination
                    break;
                } else if (d_counter[id_sequence].index[t] == index) {
                    atomicAdd(&d_counter[id_sequence].frequency[t], 1);// Value of the combination
                    break;
                }
            }
            __syncthreads();
         }
      }
   }//End for id

}

