#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include <string.h>
#include "tipos_data_struct.h"
#include "kmer_data_struct.cuh"

void GetDeviceProp(uint8_t device, lint *maxGridSize, lint *maxThreadDim, lint *deviceMemory)
{
   hipDeviceProp_t prop;

   hipGetDeviceProperties(&prop, device);

   *maxThreadDim = prop.maxThreadsDim[0];
   *maxGridSize = prop.maxGridSize[0];
   *deviceMemory = prop.totalGlobalMem;
}

void kmer_main(struct chunk *rd, lint n_concat_sequence_length, lint n_sequence, int k, ushort device)
{

   char *d_Seq;// Seq matrix
   int *d_start;
   int *d_length;// The beginning and the length of each sequence
   lint block[4], grid[4];// Grid config; 0:n_concat_sequence_length, 1:n_sequence
   lint maxGridSize, maxThreadDim, deviceMemory;// Device config
   ushort offset[4] = {1,1,1,1};
   size_t size[5], totalsize;
   int sizeOfAllCounters = 0; //
   struct counter *reads;
   int i, j;

   hipSetDevice(device);
   GetDeviceProp(device, &maxGridSize, &maxThreadDim, &deviceMemory);

//---------------------------------------------------------------------------
   size[0] = n_concat_sequence_length * sizeof(char);// d_Seq and Seq size
   size[1] = n_sequence * sizeof(int);  // d_length
   size[2] = n_sequence * sizeof(lint); // d_start
   size[3] = n_sequence * sizeof(struct counter); // d_counter
   totalsize = size[0] + (size[1] * 2) + size[2] + size[3];

   if (totalsize > deviceMemory)
   {
      printf("\n\n\t\t\t[Error] There is no enough space on GPU memory\n");
      printf("\t\t\t[Error] Required memory: %ld; Available memory: %ld\n", totalsize, deviceMemory);
      exit(1);
   }
//---------------------------------------------------------------------------

   if ( hipMalloc ((void**)&d_Seq, size[0])    != hipSuccess ) printf("\n[Error 1] %s\n", hipGetErrorString(hipGetLastError()));
   if ( hipMalloc ((void**)&d_length, size[1]) != hipSuccess ) printf("\n[Error 2] %s\n", hipGetErrorString(hipGetLastError()));
   if ( hipMalloc ((void**)&d_start, size[2])  != hipSuccess ) printf("\n[Error 3] %s\n", hipGetErrorString(hipGetLastError()));
   if ( hipMalloc ((void**)&d_n_combination, sizeof(int)) != hipSuccess )
   if ( hipMalloc ((void**)&d_counter, size[3]) != hipSuccess )  printf("\n[Error 4] %s\n", hipGetErrorString(hipGetLastError()));

   int d;
   for(d = 0; d < *rd->n_combination; d++){
      if( hipMalloc((void**)&d_counter[d].index, sizeof(int)) != hipSuccess ) printf("\n[Error 7-%d-%d] %s\n", &i, &d, hipGetErrorString(hipGetLastError()));
      if( hipMalloc((void**)&d_counter[d].frequence, sizeof(int)) != hipSuccess )  printf("\n[Error 8-%d-%d] %s\n", &i, &d, hipGetErrorString(hipGetLastError()));
   }

  
//************************************************
   // Thread mapping for raw data
   block[0] = maxThreadDim;
   grid[0] = floor(n_concat_sequence_length / block[0]) + 1;
   if (grid[0] > maxGridSize)
   {
      grid[0] = maxGridSize;
      offset[0] = (n_concat_sequence_length / (grid[0] * block[0])) + 1;
   }

   // Thread mapping for 
   block[1] = maxThreadDim;
   grid[1] = (n_sequence / block[1]) + 1;
   if (grid[1] > maxGridSize)
   {
      grid[1] = maxGridSize;
      offset[1] = (n_sequence / (grid[1] * block[1])) + 1;
   }

   block[2] = maxThreadDim;
   grid[2] = n_sequence;
   if (n_sequence > maxGridSize)
   {
      grid[2] = maxGridSize;
      offset[2] = (n_sequence / grid[2]) + 1;
   }

   block[3] = maxThreadDim;
   grid[3] = ((n_sequence*POW(k))/1024)+1;
   if (grid[3] > maxGridSize)
   {
      grid[3] = maxGridSize;
      offset[3] = (sizeOfAllCounters / (grid[3] * block[3])) + 1;
   }

//************************************************

   if ( hipMemcpyAsync(d_Seq, rd->data, size[0], hipMemcpyHostToDevice) != hipSuccess)      printf("[Error 9] %s\n", hipGetErrorString(hipGetLastError()));
   if ( hipMemcpyAsync(d_length, rd->length, size[1], hipMemcpyHostToDevice) != hipSuccess) printf("[Error 10] %s\n", hipGetErrorString(hipGetLastError()));
   if ( hipMemcpyAsync(d_start, rd->start, size[2], hipMemcpyHostToDevice) != hipSuccess)   printf("[Error 11] %s\n", hipGetErrorString(hipGetLastError()));
   if ( hipMemcpyAsync(d_reads, rd->reads, size[3], hipMemcpyHostToDevice) != hipSuccess)   printf("[Error 12] %s\n", hipGetErrorString(hipGetLastError()));

   for (i = 0; i < n_sequence; i++)
   {
      int d;
      if ( hipMemcpyAsync(d_reads[i].n_combination, rd->reads[i].n_combination, sizeof(int), hipMemcpyHostToDevice) != hipSuccess)                         printf("\n[Error 13-%d] %s\n", &i, hipGetErrorString(hipGetLastError()));
      if ( hipMemcpyAsync(d_reads[i].counter, rd->reads[i].counter, sizeof(struct counter)**d_reads[i].n_combination, hipMemcpyHostToDevice) != hipSuccess) printf("\n[Error 14-%d] %s\n", &i, hipGetErrorString(hipGetLastError()));

      for(d = 0; d < *d_reads[i].n_combination; d++){
         if ( hipMemcpyAsync(d_reads[i].counter[d].index, rd->reads[i].counter[d].index, sizeof(int), hipMemcpyHostToDevice) != hipSuccess)           printf("\n[Error 15-%d-%d] %s\n", &i, &d, hipGetErrorString(hipGetLastError()));
         if ( hipMemcpyAsync(d_reads[i].counter[d].frequence, rd->reads[i].counter[d].frequence, sizeof(int), hipMemcpyHostToDevice) != hipSuccess)   printf("\n[Error 16-%d-%d] %s\n", &i, &d, hipGetErrorString(hipGetLastError()));
      }
   }
  
//************************************************

   // SetMatrix<<<grid[0], block[0]>>>(d_counter, offset[0], n_concat_sequence_length);
   ComputeFrequence<<<n_sequence, d_length[i]>>>(d_Seq, d_reads[i], d_start[i], d_length[i], k, n_concat_sequence_length, offset[0]);


//************************************************

   for (i = 0; i < n_sequence; i++)
   {
      int d;
      for(d = 0; d < d_reads[i].n_combination; d++){
         if ( hipMemcpy(rd->reads[i].counter[d].index, d_reads[i].counter[d].index, sizeof(int), hipMemcpyHostToDevice) != hipSuccess)           printf("\n[Error 17-%d-%d] %s\n", &i, &d, hipGetErrorString(hipGetLastError()));
         if ( hipMemcpy(rd->reads[i].counter[d].frequence, d_reads[i].counter[d].frequence, sizeof(int), hipMemcpyHostToDevice) != hipSuccess)   printf("\n[Error 18-%d-%d] %s\n", &i, &d, hipGetErrorString(hipGetLastError()));
      }
   }

//************************************************

   hipFree(d_Seq);
   hipFree(d_counter);
   hipFree(d_start);
   hipFree(d_length);
   for (i = 0; i < n_sequence; i++)
   {
      int d;
      for(d = 0; d < d_reads[i].n_combination; d++){
         hipFree(d_reads[i].counter[d].index);
         hipFree(d_reads[i].counter[d].frequence);
      }
      hipFree(d_reads[i].counter);
   }
   hipFree(d_reads);

//---------------------------------------------------------------------------

   //printf("\nFim kmer_main\n");
}
