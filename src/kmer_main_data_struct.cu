#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include <string.h>
#include "tipos_data_struct.h"
#include "kmer_data_struct.cuh"

void GetDeviceProp(uint8_t device, lint *maxGridSize, lint *maxThreadDim, lint *deviceMemory) {
    hipDeviceProp_t prop;

    hipGetDeviceProperties(&prop, device);

    *maxThreadDim = prop.maxThreadsDim[0];
    *maxGridSize = prop.maxGridSize[0];
    *deviceMemory = prop.totalGlobalMem;
}

void kmer_main(struct chunk *rd, lint n_concat_sequence_length, lint n_sequence, int k, ushort device) {
    lint block[2], grid[2];// Grid config; 0:n_concat_sequence_length, 1:n_sequence
    lint maxGridSize, maxThreadDim, deviceMemory;// Device config
    ushort offset[2] = {1, 1};
    size_t size[5], totalsize;
    int sizeOfAllCounters = 0; //
    int i;
    int n_combination = *(rd->n_combination);
    hipSetDevice(device);
    GetDeviceProp(device, &maxGridSize, &maxThreadDim, &deviceMemory);
    fprintf(stderr, "Combination: %d\n", n_combination);


//---------------------------------------------------------------------------

    size[0] = n_concat_sequence_length * sizeof(char);// d_Seq and Seq size
    size[1] = n_sequence * sizeof(int);  // d_length
    size[2] = n_sequence * sizeof(lint); // d_start
    size[3] = n_sequence * sizeof(struct counter); // d_reads
    size[4] = n_combination * sizeof(int);
    totalsize = size[0] + (size[1] * 2) + size[2] + size[3] + (n_sequence * (sizeof(struct counter) + (2 * size[4])));
    fprintf(stderr, "Size[0]: %d\n", size[0]);
    fprintf(stderr, "Size[1]: %d\n", size[1] * 2);
    fprintf(stderr, "Size[2]: %d\n", size[2]);
    fprintf(stderr, "Size[3]: %d\n", size[3]);
    fprintf(stderr, "Size[4]: %d\n", size[4]);
    fprintf(stderr, "part Sum: %d\n", (n_sequence * (sizeof(struct counter) + (2 * size[4]))));

    if (totalsize > deviceMemory) {
        printf("\n\n\t\t\t[Error] There is no enough space on GPU memory\n");
        printf("\t\t\t[Error] Required memory: %ld; Available memory: %ld\n", totalsize, deviceMemory);
        exit(1);
    }

//---------------------------------------------------------------------------

    fprintf(stderr, "Memory Allocation\n");
    if (hipMallocManaged(&rd->data, size[0]) != hipSuccess)
        fprintf(stderr, "\n[Error 1] %s\n", hipGetErrorString(hipGetLastError()));
    hipDeviceSynchronize();

    if (hipMallocManaged(&rd->length, size[1]) != hipSuccess)
        fprintf(stderr, "\n[Error 2] %s\n", hipGetErrorString(hipGetLastError()));
    hipDeviceSynchronize();

    if (hipMallocManaged(&rd->start, size[2]) != hipSuccess)
        fprintf(stderr, "\n[Error 3] %s\n", hipGetErrorString(hipGetLastError()));
    hipDeviceSynchronize();

    if (hipMallocManaged(&rd->counter, size[3]) != hipSuccess)
        fprintf(stderr, "\n[Error 4] %s\n", hipGetErrorString(hipGetLastError()));
    hipDeviceSynchronize();

    for (i = 0; i < n_sequence; i++) {
        if (hipMallocManaged(&(rd->counter[i].index), size[4]) != hipSuccess)
            fprintf(stderr, "\n[Error 5-%d] %s\t", i, hipGetErrorString(hipGetLastError()));

        if (hipMallocManaged(&(rd->counter[i].frequency), size[4]) != hipSuccess)
            fprintf(stderr, "[Error 6-%d] %s", i, hipGetErrorString(hipGetLastError()));

    }
    hipDeviceSynchronize();

// ************************************************
    fprintf(stderr, "Memory Mapping\n");
    // Thread mapping for raw data
    block[0] = maxThreadDim;
    grid[0] = floor(n_concat_sequence_length / block[0]) + 1;
    if (grid[0] > maxGridSize) {
        grid[0] = maxGridSize;
        offset[0] = (n_concat_sequence_length / (grid[0] * block[0])) + 1;
    }

    // Thread mapping for
    block[1] = maxThreadDim;
    grid[1] = (n_sequence / block[1]) + 1;
    if (grid[1] > maxGridSize) {
        grid[1] = maxGridSize;
        offset[1] = (n_sequence / (grid[1] * block[1])) + 1;
    }

// ************************************************
    fprintf(stderr, "Kernel Execution Matrix\n");
    for (i = 0; i < n_sequence; i++) {
        SetMatrix<<<grid[1], block[1]>>>(rd->counter, offset[1], n_combination, i);
    }
    fprintf(stderr, "Kernel Execution Compute\n");
    hipDeviceSynchronize();

    ComputeFrequency<<<grid[0], block[0]>>>(rd->data, rd->counter, rd->start, rd->length, k, n_concat_sequence_length,
                                            offset[0], n_sequence, n_combination);
    hipDeviceSynchronize();
    for (lint t = 0; t < n_sequence; t ++)
    {
        for (int q = 0; q < n_combination; q++)
        {
//               if ( chunk[k].counter[t].index[q] != -1)
            printf("Index: %d Frequency: %d\n", rd->counter[t].index[q], rd->counter[t].frequency[q]);
        }
    }
//---------------------------------------------------------------------------

    //printf("\nFim kmer_main\n");
}
