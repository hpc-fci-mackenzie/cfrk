#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include <string.h>
#include "tipos.h"
#include "kmer.cuh"

void GetDeviceProp(uint8_t device, lint *maxGridSize, lint *maxThreadDim, lint *deviceMemory)
{
   hipDeviceProp_t prop;

   hipGetDeviceProperties(&prop, device);

   *maxThreadDim = prop.maxThreadsDim[0];
   *maxGridSize = prop.maxGridSize[0];
   *deviceMemory = prop.totalGlobalMem;
}

void kmer_main(struct read *rd, lint nN, lint nS, int k, ushort device)
{

   int *d_Index;// Index vector
   char *d_Seq;// Seq matrix
   // int *Freq;
   int *d_Freq;// Frequence vector
   int fourk;// 4 power k
   lint *d_start;
   int *d_length;// The beggining and the length of each sequence
   lint block[4], grid[4];// Grid config; 0:nN, 1:nS
   lint maxGridSize, maxThreadDim, deviceMemory;// Device config
   ushort offset[4] = {1,1,1,1};
   size_t size[5], totalsize;

   d_Index =  NULL;
   d_Seq = NULL;

   fourk = POW(k);

   hipSetDevice(device);
   GetDeviceProp(device, &maxGridSize, &maxThreadDim, &deviceMemory);

//---------------------------------------------------------------------------
   size[0] = nN * sizeof(char);// d_Seq and Seq size
   size[1] = nN * sizeof(int); // d_Index and Index size
   size[2] = nS * sizeof(int);  // d_length
   size[3] = nS * fourk * sizeof(int);// Freq and d_Freq
   size[4] = nS * sizeof(lint); // d_start
   totalsize = size[0] + size[1] + (size[2] * 2) + size[3];

   if (totalsize > deviceMemory)
   {
      printf("\n\n\t\t\t[Error] There is no enough space on GPU memory\n");
      printf("\t\t\t[Error] Required memory: %ld; Available memory: %ld\n", totalsize, deviceMemory);
      exit(1);
   }
//---------------------------------------------------------------------------

   if ( hipMalloc ((void**)&d_Seq, size[0])    != hipSuccess ) printf("\n[Error 1] %s\n", hipGetErrorString(hipGetLastError()));
   if ( hipMalloc ((void**)&d_Index, size[1])  != hipSuccess ) printf("\n[Error 2] %s\n", hipGetErrorString(hipGetLastError()));
   if ( hipMalloc ((void**)&d_start, size[4])  != hipSuccess ) printf("\n[Error 3] %s\n", hipGetErrorString(hipGetLastError()));
   if ( hipMalloc ((void**)&d_length, size[2]) != hipSuccess ) printf("\n[Error 4] %s\n", hipGetErrorString(hipGetLastError()));
   if ( hipMalloc ((void**)&d_Freq, size[3])   != hipSuccess ) printf("\n[Error 5] %s\n", hipGetErrorString(hipGetLastError()));

//************************************************
   block[0] = maxThreadDim;
   grid[0] = floor(nN / block[0]) + 1;
   if (grid[0] > maxGridSize)
   {
      grid[0] = maxGridSize;
      offset[0] = (nN / (grid[0] * block[0])) + 1;
   }

   block[1] = maxThreadDim;
   grid[1] = (nS / block[1]) + 1;
   if (grid[1] > maxGridSize)
   {
      grid[1] = maxGridSize;
      offset[1] = (nS / (grid[1] * block[1])) + 1;
   }

   block[2] = maxThreadDim;
   grid[2] = nS;
   if (nS > maxGridSize)
   {
      grid[2] = maxGridSize;
      offset[2] = (nS / grid[2]) + 1;
   }

   int nF = nS*POW(k);
   block[3] = maxThreadDim;
   grid[3] = ((nS*POW(k))/1024)+1;
   if (grid[3] > maxGridSize)
   {
      grid[3] = maxGridSize;
      offset[3] = (nF / (grid[3] * block[3])) + 1;
   }

//************************************************

   if ( hipMemcpyAsync(d_Seq, rd->data, size[0], hipMemcpyHostToDevice) != hipSuccess) printf("[Error 6] %s\n", hipGetErrorString(hipGetLastError()));
   if ( hipMemcpyAsync(d_start, rd->start, size[4], hipMemcpyHostToDevice) != hipSuccess) printf("[Error 7] %s\n", hipGetErrorString(hipGetLastError()));
   if ( hipMemcpyAsync(d_length, rd->length, size[2], hipMemcpyHostToDevice) != hipSuccess) printf("[Error 8] %s\n", hipGetErrorString(hipGetLastError()));

//************************************************


   SetMatrix<<<grid[0], block[0]>>>(d_Index, offset[0], -1, nN);
   SetMatrix<<<grid[3], block[3]>>>(d_Freq, offset[3], 0, nF);
   ComputeIndex<<<grid[0], block[0]>>>(d_Seq, d_Index, k, nN, offset[0]);
   //ComputeFreq<<<grid[1], block[1]>>>(d_Index, d_Freq, d_start, d_length, offset[1], fourk, nS, nN);
   ComputeFreqNew<<<grid[2],block[2]>>>(d_Index, d_Freq, d_start, d_length, offset[2], fourk, nS);


   //hipFree(rd);

   if ( hipHostMalloc((void**)&rd->Freq, size[3]) != hipSuccess) printf("\n[Error 9] %s\n", hipGetErrorString(hipGetLastError()));
   if ( hipMemcpy(rd->Freq, d_Freq, size[3], hipMemcpyDeviceToHost) != hipSuccess) printf("\n[Error 10] %s\n", hipGetErrorString(hipGetLastError()));



//************************************************
   hipFree(d_Seq);
   hipFree(d_Freq);
   hipFree(d_Index);
   hipFree(d_start);
   hipFree(d_length);
//---------------------------------------------------------------------------

   //printf("\nFim kmer_main\n");
}
